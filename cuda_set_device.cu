#include "hip/hip_runtime.h"
#include "mpi.h"

#include "cuda_set_device.h"

extern "C" void cuda_set_device();
extern "C" void cuda_my_memset(void* ptr, int value, size_t count);

#include <stdio.h>

void cuda_set_device(){
    int rank;
    MPI_Comm shmcomm;
    MPI_Comm_split_type(MPI_COMM_WORLD, MPI_COMM_TYPE_SHARED, 0,MPI_INFO_NULL, &shmcomm);
    MPI_Comm_rank(shmcomm, &rank);

    hipSetDevice(rank/32);
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, rank/32);
    printf("Rank %d: Using device %d: %s\n", rank, rank/32, prop.name);
}

void cuda_my_memset(void* ptr, int value, size_t count){
    hipMemset(ptr, value, count);
}
